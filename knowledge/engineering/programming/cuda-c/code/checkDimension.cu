#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void checkIndex(void)
{
	printf("threadIdx: (%d, %d, %d) blockIdx: (%d, %d, %d) blockDim: (%d, %d, %d)"
		"gridDim: (%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.y, blockIdx.z,
		blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

int main(int argc, char **argv)
{
	int nElem = 6;

	dim3 block (3);
	dim3 grid ((nElem + block.x - 1) / block.x);

	printf("grid.x %d grid.y %d gird.z %d\n", grid.x, grid.y, grid.z);
	printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);

	checkIndex <<<grid, block>>> ();
	hipDeviceReset();

	return(0);
}